
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

inline void CUDA_CHECK (hipError_t err) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
        exit(-1);
    }
}

#define N 1024*1024

__global__ void calculate_pi(int* hits) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize random number state (unique for every thread in the grid)
    int seed = 0;
    int offset = 0;
    hiprandState_t curand_state;
    hiprand_init(seed, idx, offset, &curand_state);

    // Generate random coordinates within (0.0, 1.0]
    float x = hiprand_uniform(&curand_state);
    float y = hiprand_uniform(&curand_state);

    // Increment hits counter if this point is inside the circle
    if (x * x + y * y <= 1.0f) {
        atomicAdd(hits, 1);
    }
}


int main(int argc, char** argv) {
    // Allocate host and device values
    int* hits;
    hits = (int*) malloc(sizeof(int));

    int* d_hits;
    CUDA_CHECK(hipMalloc((void**) &d_hits, sizeof(int)));

    // Initialize number of hits and copy to device
    *hits = 0;
    CUDA_CHECK(hipMemcpy(d_hits, hits, sizeof(int), hipMemcpyHostToDevice));

    // Launch kernel to do the calculation
    int threads_per_block = 256;
    int blocks = (N + threads_per_block - 1) / threads_per_block;

    calculate_pi<<<blocks, threads_per_block>>>(d_hits);
    hipDeviceSynchronize();

    // Copy final result back to the host
    CUDA_CHECK(hipMemcpy(hits, d_hits, sizeof(int), hipMemcpyDeviceToHost));

    // Calculate final value of pi
    float pi_est = (float) *hits / (float) (N) * 4.0f;

    // Print out result
    std::cout << "Estimated value of pi = " << pi_est << std::endl;
    std::cout << "Relative error = " << std::abs((M_PI - pi_est) / pi_est) << std::endl;

    // Clean up
    free(hits);
    CUDA_CHECK(hipFree(d_hits));

    return 0;
}
